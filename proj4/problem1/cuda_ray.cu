/**
 * This program performs ray tracing on the GPU using CUDA.
 */
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define SPHERES 20
#define INF 2e10f
#define DIM 2048

#define rnd(x) (x * rand() / RAND_MAX)

struct Sphere
{
    float r, g, b; // red, green, and blue color components of the sphere
    float radius;  // radius of the sphere
    float x, y, z; // coordinates of the center of the sphere.

    __device__ float hit(float ox, float oy, float *n)
    {
        float dx = ox - x;
        float dy = oy - y;
        if (dx * dx + dy * dy < radius * radius)
        {
            float dz = sqrtf(radius * radius - dx * dx - dy * dy);
            *n = dz / sqrtf(radius * radius);
            return dz + z;
        }
        return -INF;
    }
};

/**
 * This function calculates the color of a pixel at (x, y) by ray tracing.
 *
 * @param s A pointer to an array of Sphere structs which represent the spheres in the scene.
 * @param x The x-coordinate of the pixel.
 * @param y The y-coordinate of the pixel.
 * @param ptr A pointer to an array where the color of the pixel will be stored.
 *
 * The color of the pixel is determined by the color of the sphere that the ray from the pixel intersects first.
 * If the ray does not intersect any sphere, the pixel is black.
 * The color is stored in the ptr array in RGBA format, with each component as an unsigned char (0-255).
 */
__global__ void kernel(Sphere *s, unsigned char *ptr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = x + y * DIM;
    float ox = (x - DIM / 2);
    float oy = (y - DIM / 2);

    float r = 0, g = 0, b = 0;
    float maxz = -INF;
    for (int i = 0; i < SPHERES; i++)
    {
        float n;
        float t = s[i].hit(ox, oy, &n);
        if (t > maxz)
        {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }

    ptr[offset * 4 + 0] = (int)(r * 255);
    ptr[offset * 4 + 1] = (int)(g * 255);
    ptr[offset * 4 + 2] = (int)(b * 255);
    ptr[offset * 4 + 3] = 255;
}

/**
 * This function writes a bitmap to a file in PPM format.
 *
 * @param bitmap A pointer to an array that contains the bitmap data. The data should be in RGBA format, with each component as an unsigned char (0-255).
 * @param xdim The width of the bitmap in pixels.
 * @param ydim The height of the bitmap in pixels.
 * @param fp A pointer to the file where the bitmap will be written.
 *
 * The PPM file format is a simple uncompressed image format that consists of a header followed by pixel data.
 * The header contains the width and height of the image and the maximum color value (255 in this case).
 * The pixel data consists of red, green, and blue color components for each pixel in the image.
 */
void ppm_write(unsigned char *bitmap, int xdim, int ydim, FILE *fp)
{
    int i, x, y;

    fprintf(fp, "P3\n");
    fprintf(fp, "%d %d\n", xdim, ydim);
    fprintf(fp, "255\n");

    for (y = 0; y < ydim; y++)
    {
        for (x = 0; x < xdim; x++)
        {
            i = x + y * xdim;
            fprintf(fp, "%d %d %d ", bitmap[4 * i], bitmap[4 * i + 1], bitmap[4 * i + 2]);
        }
        fprintf(fp, "\n");
    }
}

/**
 * This is the main function of the program. It is the entry point where the program starts execution.
 * 
 * The main function initializes the scene, performs ray tracing to calculate the color of each pixel in the image, and writes the resulting image to a file in PPM format.
 */
int main()
{
    unsigned char *bitmap;

    srand(time(NULL));

    FILE *fp = fopen("result.ppm", "w");

    Sphere *temp_s = (Sphere *)malloc(sizeof(Sphere) * SPHERES);
    for (int i = 0; i < SPHERES; i++)
    {
        temp_s[i].r = rnd(1.0f);
        temp_s[i].g = rnd(1.0f);
        temp_s[i].b = rnd(1.0f);
        temp_s[i].x = rnd(2000.0f) - 1000;
        temp_s[i].y = rnd(2000.0f) - 1000;
        temp_s[i].z = rnd(2000.0f) - 1000;
        temp_s[i].radius = rnd(200.0f) + 40;
    }

    bitmap = (unsigned char *)malloc(sizeof(unsigned char) * DIM * DIM * 4);

    Sphere *d_s;
    unsigned char *d_bitmap;
    hipMalloc((void **)&d_s, sizeof(Sphere) * SPHERES);
    hipMalloc((void **)&d_bitmap, sizeof(unsigned char) * DIM * DIM * 4);
    hipMemcpy(d_s, temp_s, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice);

    dim3 grids(DIM / 16, DIM / 16);
    dim3 threads(16, 16);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    kernel<<<grids, threads>>>(d_s, d_bitmap);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("CUDA ray tracing: %f sec\n", elapsedTime / 1000);

    hipMemcpy(bitmap, d_bitmap, sizeof(unsigned char) * DIM * DIM * 4, hipMemcpyDeviceToHost);

    ppm_write(bitmap, DIM, DIM, fp);

    fclose(fp);
    hipFree(d_s);
    hipFree(d_bitmap);
    free(bitmap);
    free(temp_s);

    return 0;
}
